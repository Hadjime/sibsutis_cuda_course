#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_RETURN(value) {							\
	hipError_t _m_cudaStat = value;						\
	if (_m_cudaStat != hipSuccess) {						\
		fprintf(stderr, "Error %s at line %d in file %s\n",			\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);	\
		exit(1);								\
	} }

__global__ void gTest(float* a){
  a[threadIdx.x+blockDim.x*blockIdx.x]=(float)(threadIdx.x+blockDim.x*blockIdx.x);
} 

int main(){
  float *da, *ha;
  int num_of_blocks=10, threads_per_block=32;
  int N=num_of_blocks*threads_per_block;
 
  ha=(float*)calloc(N, sizeof(float));
  CUDA_CHECK_RETURN(hipMalloc((void**)&da, N*sizeof(float)));
  
  hipEvent_t  start,stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start,0);
  
  gTest<<<dim3(num_of_blocks), dim3(threads_per_block)>>>(da);
 // cudaThreadSynchronize();
  //CUDA_CHECK_RETURN(cudaDeviceSynchronize());
  //CUDA_CHECK_RETURN(cudaGetLastError());
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  fprintf(stderr,"gTest took %g\n", elapsedTime);
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  CUDA_CHECK_RETURN(hipMemcpy(ha,da,N*sizeof(float), hipMemcpyDeviceToHost));
  
  for(int i=0;i<N;i++)
	  printf("%g\n", ha[i]);

  free(ha);
  hipFree(da);
  return 0;
}
