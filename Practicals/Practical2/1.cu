#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>

__global__  void gInit(float* a, float* b, int N){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    
    unsigned int seed=thread_id;
    hiprandState s;
    hiprand_init(seed,0,0, &s);  
    
    for(int i=thread_id; i<N; i+=blockDim.x*gridDim.x){
      a[i]=hiprand_uniform(&s);
      b[i]=hiprand_uniform(&s);  
    }
}

__global__ void gSum(float* a, float* b, float* c, int N){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    for(int i=thread_id; i<N; i+=blockDim.x*gridDim.x)
      c[i]=a[i]+b[i];
  }

int main(int argc, char* argv[]){
  float *a, *b, *c;
  float *ha, *hb, *hc;
  if(argc<4) { fprintf(stderr, "USAGE: 1 <vector length> <blocks>  <threads>\n"); return 1;}
  int N=atoi(argv[1]);
  int num_of_blocks=atoi(argv[2]);
  int threads_per_block=atoi(argv[3]);  
  
  hipMalloc((void**)&a, N*sizeof(float));  
  hipMalloc((void**)&b, N*sizeof(float));  
  hipMalloc((void**)&c, N*sizeof(float));

  ha=(float*)calloc(N, sizeof(float));
  hb=(float*)calloc(N, sizeof(float));
  hc=(float*)calloc(N, sizeof(float));
    
  gInit<<<num_of_blocks,threads_per_block>>>(a,b,N);
  hipDeviceSynchronize();
  
hipEvent_t  start,stop;
float elapsedTime;

hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start,0);  
  
  gSum<<<num_of_blocks,threads_per_block>>>(a,b,c,N);
//  hipDeviceSynchronize();

hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime,start,stop); 
fprintf(stderr,"gTest took %g\n", elapsedTime);
  
  hipMemcpy(ha,a,N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hb,b,N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hc,c,N*sizeof(float), hipMemcpyDeviceToHost);  
  
  for(int i=0;i<N;i++)
    printf("%g\t%g\t%g\n", ha[i],hb[i], hc[i]);
  
  hipFree(a);
  hipFree(b);
  hipFree(c);
  free(ha);
  free(hb);
  free(hc);
  
  return 0;
}
