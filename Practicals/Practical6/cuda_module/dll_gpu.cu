#include "hip/hip_runtime.h"
#include "windows.h"
#include <hip/hip_runtime.h>

BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
		
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}


__global__  void gSum_vec(float* v1, float* v2, int N){
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	v1[i]+=v2[i];
}

extern "C"{
  __declspec(dllexport)  void Sum_vec(float* v1, float* v2, int N){
	float *u1,*u2;
	hipMalloc((void **) &u1, N*sizeof(float));
	hipMalloc((void **) &u2, N*sizeof(float));

	hipMemcpy(u1, v1, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(u2, v2, N*sizeof(float), hipMemcpyHostToDevice);	
	
	gSum_vec<<<dim3(N/512+((N%512)?1:0)),dim3(512)>>>(u1,u2,N);
	hipDeviceSynchronize();  

	hipMemcpy(v1, u1, N*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(u1);
	hipFree(u2);
  }
}

extern "C"{
__declspec(dllexport) void sum_vec(float* v1, float* v2, int N){
	for(int i=0;i<N;i++)
		v1[i]+=v2[i];
  }
}