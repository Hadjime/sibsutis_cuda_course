#include "hip/hip_runtime.h"
#include <stdio.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

                            
#define N (1024*1024)
#define FULL_DATA_SIZE  (N*20)

__global__ void kernel(int* a, int* b, int* c){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<N){
    int idx1=(idx+1)%256;
    int idx2=(idx+2)%256;
    float as=(a[idx]+a[idx1]+a[idx2])/3.0f;
    float bs=(b[idx]+b[idx1]+b[idx2])/3.0f;
    c[idx]=(as+bs)/2;
  }
}

int main(){
  hipDeviceProp_t prop;
  int whichDevice;
  
  HANDLE_ERROR(
  hipGetDevice(&whichDevice)
  );
  HANDLE_ERROR(
  hipGetDeviceProperties(&prop, whichDevice)
  );
  if(!prop.deviceOverlap){
    printf("Device does not support overlapping\n");
    return 0;
  }
  
  hipEvent_t start, stop;
  float elapsedTime;
  
 /////////////////// hipStream_t stream;
  int *host_a, *host_b, *host_c;
  //int *h_c;
  int *dev_a, *dev_b, *dev_c;
  
  HANDLE_ERROR(
  hipEventCreate(&start)
  );
  HANDLE_ERROR(
  hipEventCreate(&stop)
  );
  
  
  HANDLE_ERROR(
  hipMalloc( (void**)&dev_a, N*sizeof(int)) 
  );
  HANDLE_ERROR(
  hipMalloc( (void**)&dev_b, N*sizeof(int)) 
  );
  HANDLE_ERROR(
  hipMalloc( (void**)&dev_c, N*sizeof(int)) 
  );

  
  HANDLE_ERROR(
  hipHostAlloc( (void**)&host_a, FULL_DATA_SIZE*sizeof(int), 
		 hipHostMallocDefault) 
  );
  HANDLE_ERROR(
  hipHostAlloc( (void**)&host_b, FULL_DATA_SIZE*sizeof(int),
    hipHostMallocDefault) 
  );
  HANDLE_ERROR(
  hipHostAlloc( (void**)&host_c, FULL_DATA_SIZE*sizeof(int),
  hipHostMallocDefault) 
  );
  
//h_c=(int*)calloc(FULL_DATA_SIZE, sizeof(int));  
  
  for(int i=0; i<FULL_DATA_SIZE;i++){
    host_a[i]=rand();
    host_b[i]=rand();    
  }

  hipStream_t stream;
  HANDLE_ERROR(
  hipStreamCreate(&stream)
  ); 
 
  HANDLE_ERROR(
  hipEventRecord(start,0)
  );  
  for(int i=0; i<FULL_DATA_SIZE; i+=N){
      HANDLE_ERROR(
      hipMemcpyAsync(dev_a, host_a+i, N*sizeof(int), 
		      hipMemcpyHostToDevice, stream)
      );
      HANDLE_ERROR(
      hipMemcpyAsync(dev_b, host_b+i, N*sizeof(int), 
		      hipMemcpyHostToDevice, stream)
      );
    
      kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
  
      HANDLE_ERROR(
      hipMemcpyAsync(host_c+i, dev_c, N*sizeof(int), 
		      hipMemcpyDeviceToHost, stream)
      );
  }
   
  HANDLE_ERROR( hipStreamSynchronize( stream ) );

  HANDLE_ERROR( 
  hipEventRecord(stop,0)  
  );  
  HANDLE_ERROR( 
  hipEventSynchronize(stop)  
  );  
  HANDLE_ERROR( 
  hipEventElapsedTime(&elapsedTime, start, stop)  
  );  
  printf("Elapsed time: %3.1f ms\n", elapsedTime );
  
  HANDLE_ERROR(hipHostFree(host_a));  
  HANDLE_ERROR(hipHostFree(host_b)); 
  HANDLE_ERROR(hipHostFree(host_c));
  
  HANDLE_ERROR(hipFree(dev_a));  
  HANDLE_ERROR(hipFree(dev_b)); 
  HANDLE_ERROR(hipFree(dev_c));
  
//free(h_c);
  
  HANDLE_ERROR(hipStreamDestroy(stream));
  
  return 0;
}