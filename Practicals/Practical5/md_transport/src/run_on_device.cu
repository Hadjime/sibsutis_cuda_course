#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include "common.h"
#include "kerns.h"
int RunOnDevice(REAL* Df, INPUT& input,int* assigned_devices){
////////////////// Select the used device:///////////////////////////////////////
   int used_device;
   if ( hipSetDevice(assigned_devices[0]) != hipSuccess or
        hipGetDevice( &used_device ) != hipSuccess or
        used_device != assigned_devices[0]
      ) {
        printf ("Error: unable to set device %d\n", assigned_devices[0]);
        return 1;
    }
    printf ("Used device: %d\n", used_device);
/////////////////////////////////////////////////////////////////////////////////
    if(input.M>8 || input.L> pow(2,15)){
      fprintf(stderr,"Dimension is out of size\n"); 
      return 2; 
    }
    int size_of_df=input.M*input.M*input.M*input.L*sizeof(REAL);    

    REAL hv=(input.vmax-input.vmin)/input.M;
    REAL h=(input.xmax-input.xmin)/input.L;

    REAL *df_device[NGPUS];
    
    hipStream_t mdStream[NGPUS];
    hipEvent_t  mdEvent[NGPUS];
  ///////////////MEMORY ALLOCATION//////////////////////////////////////////
    for(int idev=0;idev<NGPUS;idev++){
      hipSetDevice(assigned_devices[idev]);
      hipStreamCreate(&mdStream[idev]);
      hipEventCreate(&mdEvent[idev]);
      hipMalloc((void **) &df_device[idev], size_of_df/NGPUS);
    }
  
    for(int idev=0;idev<NGPUS;idev++){
      hipSetDevice(assigned_devices[idev]);
      gInitializeDf<<<dim3(input.L/NGPUS),dim3(input.M,input.M,input.M)>>>(idev, df_device[idev], input.vmin, hv, h);
      hipEventRecord( mdEvent[idev], mdStream[idev] ); //асинхронный вызов ядер
    }
    for(int idev=0;idev<NGPUS;idev++){
      hipEventSynchronize(mdEvent[idev]); //синхронизация выполнения ядер
    }
    
    for(int idev=0;idev<NGPUS;idev++){
      hipSetDevice(assigned_devices[idev]);
      hipMemcpy(Df+idev*input.L*input.M*input.M*input.M/NGPUS, 
      df_device[idev], size_of_df/NGPUS, hipMemcpyDeviceToHost);
      hipEventRecord( mdEvent[idev], mdStream[idev] );
   }
    for(int idev=0;idev<NGPUS;idev++){
      hipEventSynchronize(mdEvent[idev]);
    }
   
    
    for(int l=0;l<input.L;l++)
    for(int k=0;k<input.M;k++)
      for(int j=0;j<input.M;j++)
	for(int i=0;i<input.M;i++)
	  std::cout<<Df[i + j*input.M + k*input.M*input.M + l*input.M*input.M*input.M]<<std::endl;    
    
    
    
    for(int idev=0;idev<NGPUS;idev++){
      hipSetDevice(assigned_devices[idev]);
      hipFree(df_device[idev]);
    }
    
    return 0;
}
