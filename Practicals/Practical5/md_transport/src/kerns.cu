#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
__global__  void gInitializeDf(int idev, REAL* df, REAL vmin, REAL hv, REAL h){
  int i=threadIdx.x;//+blockIdx.x*blockDim.x;
  int j=threadIdx.y;//+blockIdx.y*blockDim.y;
  int k=threadIdx.z;//+blockIdx.z*blockDim.z;
  
  int x=idev*gridDim.x+blockIdx.x;
  int l=blockIdx.x;
  int M=blockDim.x;
  int L=gridDim.x*NGPUS;
  
  REAL V=0.1;
  REAL T=1.0;
  REAL Rho=10.0*exp(-(x-L/2.0)*(x-L/2.0)/0.1);
  
  REAL vx=vmin+(i+0.5)*hv;
  REAL vy=vmin+(j+0.5)*hv;
  REAL vz=vmin+(k+0.5)*hv;  
  REAL dummy=df[i+j*M+k*M*M +l*M*M*M]=Rho*exp( -( (vx-V)*(vx-V) + vy*vy + vz*vz )/2.0/T )/pow(2.0*pi*T,1.5);
  printf("%d\t%d\t%d\t%d\t%g\n",i,j,k,l,dummy);
}
