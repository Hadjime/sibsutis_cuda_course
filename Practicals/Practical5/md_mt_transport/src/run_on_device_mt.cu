#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include "common.h"
#include "kerns.h"
int RunOnDevice(REAL* Df, INPUT& input,int* assigned_devices){
////////////////// Select the used device:///////////////////////////////////////
   int used_device;
   if ( hipSetDevice(assigned_devices[0]) != hipSuccess or
        hipGetDevice( &used_device ) != hipSuccess or
        used_device != assigned_devices[0]
      ) {
        printf ("Error: unable to set device %d\n", assigned_devices[0]);
        return 1;
    }
    printf ("Used device: %d\n", used_device);
/////////////////////////////////////////////////////////////////////////////////
    if(input.M>8 || input.L> pow(2,15)){
      fprintf(stderr,"Dimension is out of size\n"); 
      return 2; 
    }
    int size_of_df=input.M*input.M*input.M*input.L*sizeof(REAL);    

    REAL hv=(input.vmax-input.vmin)/input.M;
    REAL h=(input.xmax-input.xmin)/input.L;

    REAL *df_device[NGPUS];
    
    omp_set_num_threads(NGPUS); 
    #pragma omp parallel
    {
      unsigned int idev = omp_get_thread_num();
      hipSetDevice(assigned_devices[idev]);
      hipMalloc((void **) &df_device[idev], size_of_df/NGPUS);
      gInitializeDf<<<dim3(input.L/NGPUS),dim3(input.M,input.M,input.M)>>>(idev, df_device[idev], input.vmin, hv, h);
      hipDeviceSynchronize();
      hipMemcpy(Df+idev*input.L*input.M*input.M*input.M/NGPUS, 
		 df_device[idev], size_of_df/NGPUS, hipMemcpyDeviceToHost);
      hipFree(df_device[idev]);
    }
   
    return 0;
}
