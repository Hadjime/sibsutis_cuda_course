#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <stdio.h>

#define REAL float

__global__ void gInit(REAL* v){
  v[threadIdx.x+blockIdx.x*blockDim.x]=0.1F*(threadIdx.x+blockIdx.x*blockDim.x);
 }

int main(int argc, char* argv[]){
  REAL *v1, *v2;

int num_of_blocks=atoi(argv[1]);
int threads_per_block=atoi(argv[2]);

  int N=threads_per_block*num_of_blocks;

  hipMalloc((void**)&v1, N*sizeof(REAL));
  hipMalloc((void**)&v2, N*sizeof(REAL));



  gInit<<<num_of_blocks, threads_per_block>>>(v1);
  hipDeviceSynchronize();

  gInit<<<num_of_blocks,threads_per_block >>>(v2);
  hipDeviceSynchronize();

  thrust::device_ptr<float> v1_ptr = thrust::device_pointer_cast(v1);
  thrust::device_ptr<float> v2_ptr = thrust::device_pointer_cast(v2);
  
hipEvent_t  start,stop;
float elapsedTime;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start,0);

  REAL s=thrust::inner_product(v1_ptr, v1_ptr+N, v2_ptr,0.0);
  fprintf(stderr,"Scalar Product (thrust) s =%g\n",s);

hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime,start,stop);
fprintf(stderr,"Thrust Reduce: %g\n",elapsedTime);						  
hipEventDestroy(start);
hipEventDestroy(stop);	

 return 0;
}