#include "hip/hip_runtime.h"
#define REAL float
#define SHMEM_SIZE  512

#include <stdio.h>

__global__ void gScalarProduct(REAL* v1, REAL* v2, int N, REAL* S){
  __shared__ float s[SHMEM_SIZE];
  
  int i=threadIdx.x;//+blockIdx.x*blockDim.x;
  
  int i_center;
  
  s[i]=v1[threadIdx.x+blockIdx.x*blockDim.x]*v2[threadIdx.x+blockIdx.x*blockDim.x];
  __syncthreads();
  //printf("s=%g\n", s[i]);

  i_center=blockDim.x/2;
  
  while(i_center!=0){
    if(i<i_center)
       s[i]+=s[i+i_center];
  
    __syncthreads();
    i_center/=2;
  }
  
  if(threadIdx.x==0){
    atomicAdd(S,s[0]);
    //printf("S=%g\ts[0]=%g\n",*S, s[0]);    
}
 
}


__global__ void gInit(REAL* v){
  v[threadIdx.x+blockIdx.x*blockDim.x]=0.1F*(threadIdx.x+blockIdx.x*blockDim.x);
 }


int main(int argc, char* argv[]){

int num_of_blocks=atoi(argv[1]);
int threads_per_block=atoi(argv[2]);

int N=threads_per_block*num_of_blocks;

REAL *v1, *v2;
REAL* pS_d;
REAL S_h=0;

hipMalloc((void**)&v1, N*sizeof(REAL));
hipMalloc((void**)&v2, N*sizeof(REAL));
hipMalloc((void**)&pS_d, sizeof(REAL));

  gInit<<<num_of_blocks, threads_per_block>>>(v1);
  hipDeviceSynchronize();

  gInit<<<num_of_blocks,threads_per_block >>>(v2);
  hipDeviceSynchronize();
 
  hipMemcpy(pS_d,&S_h, sizeof(REAL), hipMemcpyHostToDevice); 

hipEvent_t  start,stop;
float elapsedTime;
hipEventCreate(&start);
hipEventCreate(&stop); 

hipEventRecord(start,0);

  gScalarProduct<<<num_of_blocks, threads_per_block>>>(v1, v2, N, pS_d);
  hipDeviceSynchronize();

  hipMemcpy(&S_h, pS_d,sizeof(REAL), hipMemcpyDeviceToHost);

  printf("Scalar product is equal to %g\n", S_h);
		
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime,start,stop);
fprintf(stderr,"gScalarProduct took: %g\n",elapsedTime);
hipEventDestroy(start);
hipEventDestroy(stop);							  

 hipFree(v1);
 hipFree(v2);
 hipFree(pS_d);
 return 0;
}