#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>
__global__ void gTest(int* d){
  
  printf("%d\n", d[threadIdx.x]);
}
int main(){
size_t N = 32;
// raw pointer to device memory
int * raw_ptr;
hipMalloc((void **) &raw_ptr, N * sizeof(int));
// wrap raw pointer with a device_ptr
thrust::device_ptr<int> dev_ptr(raw_ptr);
// use device_ptr in thrust algorithms
thrust::fill(dev_ptr, dev_ptr + N, (int) 23);

//raw_ptr = thrust::raw_pointer_cast(dev_ptr);

 gTest<<<2,16>>>(raw_ptr); 
 hipDeviceSynchronize();
 return 0;
}

