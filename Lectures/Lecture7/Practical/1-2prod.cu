#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
//#include <thrust/sort.h>
#include <thrust/copy.h>
//#include <cstdlib>
#include <cstdio>
__global__ void gTest(float* d){
  int idx=threadIdx.x+blockDim.x*blockIdx.x;
  d[idx]+=(float)idx;
}

int main(void){
  float *raw_ptr;

  thrust::host_vector<float> h(1 << 8);
  thrust::fill(h.begin(), h.end(), 3.1415f);

  thrust::device_vector<float> d = h;

  raw_ptr = thrust::raw_pointer_cast(&d[0]);//d.data());

  gTest<<<4,64>>>(raw_ptr); 
  hipDeviceSynchronize();

  thrust::copy(d.begin(), d.end(), h.begin());
  
  for(int i=0;i<(1<<8);i++)
    printf("%g\n",h[i]);

  return 0;
}