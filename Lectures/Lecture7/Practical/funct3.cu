#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
//#include <thrust/generate.h>
#include <thrust/execution_policy.h>
#include <cstdio>
//#include <thrust/for_each.h>
#include <cmath>

struct range_functor
{
  float h;
  range_functor(float _h):h(_h){}
  __host__ __device__
  float operator()(float x){
    return h*x;
  }
};
struct sin_functor
{
  __host__ __device__
  float operator()(float x){
    return sin(x);
  }
};

int main(){
  range_functor rfunc(0.02);
  sin_functor  sfunc;

  thrust::host_vector<float> h1(1 << 8);
  thrust::host_vector<float> h2(1 << 8);
  thrust::device_vector<float> d1(1 << 8);// = h1;  
  thrust::device_vector<float> d2(1 << 8);// = h2;  
  thrust::sequence(thrust::device,d1.begin(), d1.end());
  thrust::transform(d1.begin(), d1.end(), d1.begin(), rfunc);  
  thrust::transform(d1.begin(), d1.end(), d2.begin(), sfunc);  

  h2=d2;
  h1=d1;
 //thrust::for_each(thrust::device, d2.begin(), d2.end(), printf_functor()); 
    for(int i=0;i<(1<<8);i++){
    printf("%g\t%g\n",h1[i], h2[i]);
  }
  
  
  return 0;
}
