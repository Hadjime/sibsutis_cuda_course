#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <stdio.h>

__global__ void gTest(int* d){
  
  printf("%d\n", d[threadIdx.x+blockDim.x*blockIdx.x]);
}

int main(){
size_t N = 32;
int * raw_ptr;

thrust::device_ptr<int> dev_ptr = thrust::device_malloc<int>(N);
// use device_ptr in thrust algorithms
thrust::sequence(dev_ptr, dev_ptr + N, (int) 523);

raw_ptr = thrust::raw_pointer_cast(dev_ptr);

 gTest<<<2,16>>>(raw_ptr); 
 hipDeviceSynchronize();
 return 0;
}
