#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <cstdlib>
int main(void)
{
  float *h, *d;
  hipHostMalloc((void**)&h, 1<<24);
  hipMalloc((void**)&d, 1<<24);
  hipMemcpyAsync(h,d,2<<10,  hipMemcpyHostToDevice);
// generate 16M random numbers on the host
thrust::host_vector<int> h_vec(1 << 24);
thrust::generate(h_vec.begin(), h_vec.end(), rand);
// transfer data to the device
thrust::device_vector<int> d_vec = h_vec;
// transfer data back to host
thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

hipHostFree(h);
hipFree(d);
return 0;
}
